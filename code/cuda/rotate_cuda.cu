#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define PI 3.1415926 // Define the value of pi
#define FILENAME "im.pgm" // Define the name of the PGM image file

// Define a structure to store information about the PGM image
typedef struct {
    int width; // Width of the image
    int height; // Height of the image
    int maxval; // Maximum grayscale value of the image
    unsigned char* data; // Pixel data of the image stored in a one-dimensional array
} PGMImage;

// Function to read a PGM image file and store its information in a PGMImage structure
void readPGM(PGMImage* image, const char* filename) {
    FILE* fp = fopen(filename, "r"); // Open the file in text mode
    if (fp == NULL) { // If the file opening fails, print an error message and exit the program
        perror("Cannot open file to read");
        exit(EXIT_FAILURE);
    }

    char ch; // Variable to store characters from the file
    int i; // Loop counter

    // Read the first line of the file and check if it is the P2 identifier
    if (fscanf(fp, "%c%c", &ch, &ch) != 2) {
        fprintf(stderr, "Error reading file header\n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }
    if (ch != '2') { // If it is not the P2 identifier, print an error message and exit the program
        fprintf(stderr, "Not a valid P2 PGM file\n");
        exit(EXIT_FAILURE);
    }

    // Skip the newline character after the first line of the file
    fgetc(fp);

    // Skip comment lines in the file, if any
    while ((ch = fgetc(fp)) == '#') {
        while (fgetc(fp) != '\n');
    }

    // Put the last read character back into the file stream for later use with fscanf
    ungetc(ch, fp);

    // Read the width, height, and maximum grayscale value of the image from the file
    if (fscanf(fp, "%d%d%d", &image->width, &image->height, &image->maxval) != 3) {
        fprintf(stderr, "Error reading image size and maxval\n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Dynamically allocate memory for storing the pixel data based on the width and height of the image
    image->data = (unsigned char*)malloc(image->width * image->height * sizeof(unsigned char));
    if (image->data == NULL) {
        fprintf(stderr, "Failed to allocate memory for pixel data\n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Read the pixel data of the image from the file and store it in a one-dimensional array
    for (i = 0; i < image->width * image->height; i++) {
        if (fscanf(fp, "%hhu", &image->data[i]) != 1) {
            fprintf(stderr, "Error reading pixel data\n");
            fclose(fp);
            free(image->data);
            exit(EXIT_FAILURE);
        }
    }

    // Close the file
    fclose(fp);
}


// Function to write a PGM image file, writing the information from a PGMImage structure to the file
void writePGM(PGMImage* image, const char* filename) {
    FILE* fp = fopen(filename, "w"); // Open the file in text mode
    if (fp == NULL) { // If the file opening fails, print an error message and exit the program
        perror("Cannot open file to write");
        exit(EXIT_FAILURE);
    }

    int i; // Loop counter

    // Write the P2 identifier to the first line of the file
    fprintf(fp, "P2\n");

    // Write the width and height of the image to the second line of the file
    fprintf(fp, "%d %d\n", image->width, image->height);

    // Write the maximum grayscale value of the image to the third line of the file
    fprintf(fp, "%d\n", image->maxval);

    // Write the pixel data of the image to the file, separating each value with a space
    for (i = 0; i < image->width * image->height; i++) {
        fprintf(fp, "%hhu ", image->data[i]);
    }

    // Close the file
    fclose(fp);
}

__global__ void rotatePGMKernel(unsigned char* src, unsigned char* dst, int srcWidth, int srcHeight, int dstWidth, int dstHeight, double cosine, double sine, double ori_centre_x, double ori_centre_y, double new_centre_x, double new_centre_y) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dstWidth && y < dstHeight) {
        double sx = (y - new_centre_y) * sine + (x - new_centre_x) * cosine + ori_centre_x;
        double sy = (y - new_centre_y) * cosine - (x - new_centre_x) * sine + ori_centre_y;

        if (sx >= 0 && sx < srcWidth && sy >= 0 && sy < srcHeight) {
            int x1 = (int)(sx + 0.5);
            int y1 = (int)(sy + 0.5);
            dst[y * dstWidth + x] = src[y1 * srcWidth + x1];
        } else {
            dst[y * dstWidth + x] = 0;
        }
    }
}

void rotatePGM(PGMImage* src, PGMImage* dst, double angle) {
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing the rotatePGM function
    hipEventRecord(start);

    double radian = angle * PI / 180;
    double cosine = cos(radian);
    double sine = sin(radian);

    double ori_centre_x = (src->width - 1) / 2.0;
    double ori_centre_y = (src->height - 1) / 2.0;

    dst->height = (int)ceil(fabs(src->height * cosine) + fabs(src->width * sine));
    dst->width = (int)ceil(fabs(src->width * cosine) + fabs(src->height * sine));

    double new_centre_x = (dst->width - 1) / 2.0;
    double new_centre_y = (dst->height - 1) / 2.0;

    dst->maxval = src->maxval;
    dst->data = (unsigned char*)malloc(dst->width * dst->height * sizeof(unsigned char));

    unsigned char *d_src, *d_dst;
    size_t srcSize = src->width * src->height * sizeof(unsigned char);
    size_t dstSize = dst->width * dst->height * sizeof(unsigned char);

    hipMalloc(&d_src, srcSize);
    hipMalloc(&d_dst, dstSize);

    hipMemcpy(d_src, src->data, srcSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((dst->width + 31) / 32, (dst->height + 31) / 32);
    rotatePGMKernel<<<numBlocks, threadsPerBlock>>>(d_src, d_dst, src->width, src->height, dst->width, dst->height, cosine, sine, ori_centre_x, ori_centre_y, new_centre_x, new_centre_y);

    hipMemcpy(dst->data, d_dst, dstSize, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dst);

    // Stop timing the rotatePGM function
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time for rotatePGM function
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("rotatePGM execution time: %f ms\n", milliseconds);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


int main() {
    // Create CUDA events for timing the entire program
    hipEvent_t startProgram, stopProgram;
    hipEventCreate(&startProgram);
    hipEventCreate(&stopProgram);

    // Start timing the entire program
    hipEventRecord(startProgram);

    PGMImage src, dst;
    double angle = 45; // Example rotation angle, replace with desired value or argument

    // Read the source image
    readPGM(&src, FILENAME);

    // Perform the rotation
    rotatePGM(&src, &dst, angle);

    // Write the rotated image to a file
    writePGM(&dst, "rotated_im.pgm");

    // Free allocated memory
    free(src.data);
    free(dst.data);

    // Stop timing the entire program
    hipEventRecord(stopProgram);
    hipEventSynchronize(stopProgram);

    // Calculate and print the elapsed time for the entire program
    float totalMilliseconds = 0;
    hipEventElapsedTime(&totalMilliseconds, startProgram, stopProgram);
    printf("Total program execution time: %f ms\n", totalMilliseconds);

    // Clean up CUDA events
    hipEventDestroy(startProgram);
    hipEventDestroy(stopProgram);

    return 0;
}





